#include "hip/hip_runtime.h"
/**
 * University of Pittsburgh
 * Department of Computer Science
 * CS1645: Introduction to HPC Systems
 * Instructor: Xiaolong Cui
 * This is a skeleton for implementing prefix sum using GPU, inspired
 * by nvidia course of similar name.
 */

#include <stdio.h>
#include <math.h>
#include <string.h>

#define N 512

/*
 * You should implement the parallel scan function here!
 */
__global__ void parallel_scan(float *g_odata, float *g_idata, int n) {
	g_odata[threadIdx.x] = 0.0;
	extern  __shared__  float temp[];
	int thread_id = threadIdx.x;
	int p_out = 0, p_in = 1;
	 temp[p_out * n + thread_id] = (thread_id > 0) ? g_idata[thread_id - 1] : 0; 
	__syncthreads();
	for(int i = 1; i < n; i *= 2) {
		p_out = 1 - p_out;
		p_in = 1 - p_out;
		if(thread_id >= i){
			temp[p_out * n + thread_id] = temp[p_in * n + thread_id - i] + temp[p_in * n + thread_id];
		}
		else{
			temp[p_out* n + thread_id] = temp[p_in * n + thread_id];
		}
		__syncthreads();
	}
	g_odata[thread_id] = temp[p_out * n + thread_id];
}
/*
 * Fills an array a with n random floats.
 */
void random_floats(float* a, int n) {
  float d;
  // Comment out this line if you want consistent "random".
  srand(time(NULL));
  for (int i = 0; i < n; ++i) {
    d = rand() % 8;
    a[i] = ((rand() % 64) / (d > 0 ? d : 1));
  }
}
/*
 * Simple Serial implementation of exclusive scan.
 */
void serial_scan(float* out, float* in, int n) {
  float total_sum = 0;
  out[0] = 0;
  for (int i = 1; i < n; i++) {
    total_sum += in[i-1];
    out[i] = out[i-1] + in[i-1];
  }
  if (total_sum != out[n-1]) {
    printf("Warning: exceeding accuracy of float.\n");
  }
}
/*
 * This is a simple function that confirms that the output of the scan
 * function matches that of a golden image (array).
 */
bool printError(float *gold_out, float *test_out, bool show_all) {
  bool firstFail = true;
  bool error = false;
  float epislon = 0.1;
  float diff = 0.0;
  for (int i = 0; i < N; ++i) {
    diff = abs(gold_out[i] - test_out[i]);
    if ((diff > epislon) && firstFail) {
      printf("ERROR: gold_out[%d] = %f != test_out[%d] = %f // diff = %f \n", i, gold_out[i], i, test_out[i], diff);
      firstFail = show_all;
      error = true;
    }
  }
  return error;
}
int main(void) {
	float *in, *out, *gold_out; // host
	float *devIn, *devOut;
	int size = sizeof(float) * N;
	hipMalloc((void **)&devIn, size);
	hipMalloc((void **)&devOut, size);
	in = (float *)malloc(size);
	random_floats(in, N);
	out = (float *)malloc(size);
	gold_out = (float *)malloc(size);

  // ***********
  // RUN SERIAL SCAN
  // ***********
  serial_scan(gold_out, in, N);

  // ***********
  // RUN PARALLEL SCAN
  // ***********
	parallel_scan<<< 1, 512, N * 2 * sizeof(float)>>>(devOut, devIn, N);
	hipDeviceSynchronize();
	hipMemcpy(out, devOut, size, hipMemcpyDeviceToHost);
  if (printError(gold_out, out, false)) {
	printf("ERROR: The parallel scan function failed to produce proper output.\n");
  } else {
	printf("CONGRATS: The parallel scan function produced proper output.\n");
  }
  return 0;
}
